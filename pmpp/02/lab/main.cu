#include <cmath>
#include <stdio.h>
#include "support.h"
#include "kernel.cu"

void vector_add_cpu(float *a, float *b, float *c, int n) {
    for (int i = 0; i < n; i++) {
        c[i] = a[i] + b[i];
    }
}

int main(int argc, char**argv) {

    Timer timer;
    hipError_t cuda_ret;

    // Initialize host variables ----------------------------------------------

    printf("\nSetting up the problem..."); fflush(stdout);
    startTime(&timer);

    unsigned int n;
    if(argc == 1) {
        n = 10000;
    } else if(argc == 2) {
        n = atoi(argv[1]);
    } else {
        printf("\n    Invalid input parameters!"
           "\n    Usage: ./vecadd               # Vector of size 10,000 is used"
           "\n    Usage: ./vecadd <m>           # Vector of size m is used"
           "\n");
        exit(0);
    }

    float* A_h = (float*) malloc( sizeof(float)*n );
    for (unsigned int i=0; i < n; i++) { A_h[i] = (rand()%100)/100.00; }

    float* B_h = (float*) malloc( sizeof(float)*n );
    for (unsigned int i=0; i < n; i++) { B_h[i] = (rand()%100)/100.00; }

    float* C_h = (float*) malloc( sizeof(float)*n );

    stopTime(&timer); printf("%f s\n", elapsedTime(timer));
    printf("    Vector size = %u\n", n);

    // Allocate device variables ----------------------------------------------

    printf("Allocating device variables..."); fflush(stdout);
    startTime(&timer);

    float *A_d; 
    float *B_d; 
    float *C_d;
    int size = n * sizeof(float);
    hipMalloc((void **) &A_d, size);
    hipMalloc((void **) &B_d, size);
    hipMalloc((void **) &C_d, size);


    cuda_ret = hipDeviceSynchronize();
	if(cuda_ret != hipSuccess) FATAL("Unable to allocate device memory");
    stopTime(&timer); printf("%f s\n", elapsedTime(timer));

    // Copy host variables to device ------------------------------------------

    printf("Copying data from host to device..."); fflush(stdout);
    startTime(&timer);

    hipMemcpy(A_d, A_h, size, hipMemcpyHostToDevice);
    hipMemcpy(B_d, B_h, size, hipMemcpyHostToDevice);

    cuda_ret = hipDeviceSynchronize();
	if(cuda_ret != hipSuccess) FATAL("Unable copy data from host to device");
    stopTime(&timer); printf("%f s\n", elapsedTime(timer));

    // Launch kernel ----------------------------------------------------------

    printf("Launching kernel..."); fflush(stdout);
    startTime(&timer);

    //INSERT CODE HERE

    int threadsPerBlock = 256;
    int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock; // Integer ceiling division
    vecAddKernel <<< blocksPerGrid, threadsPerBlock >>> (A_d, B_d, C_d, n);
    
    cuda_ret = hipDeviceSynchronize();
	if(cuda_ret != hipSuccess) FATAL("Unable to launch kernel");
    stopTime(&timer); printf("%f s\n", elapsedTime(timer));

    // Copy device variables from host ----------------------------------------

    printf("Copying data from device to host..."); fflush(stdout);
    startTime(&timer);

    //INSERT CODE HERE
    hipMemcpy(C_h, C_d, size, hipMemcpyDeviceToHost);

    hipDeviceSynchronize();
    stopTime(&timer); printf("%f s\n", elapsedTime(timer));

    // Verify correctness -----------------------------------------------------
    // verify(A_h, B_h, C_h, n);
    bool correct = true;
    for(int i = 0; i < n; i++) {
        float sum = A_h[i] + B_h[i];
        float relativeError = (sum - C_h[i])/sum;
        if (relativeError > 1e-6
          || relativeError < -1e-6) {
          correct = false;
          break;
        }
    }
    if (correct) {
        printf("TEST PASSED\n\n");
    } else {
        printf("TEST FAILED\n\n");
    }
    // Free memory ------------------------------------------------------------

    free(A_h);
    free(B_h);
    free(C_h);

    //INSERT CODE HERE
    hipFree(A_d);
    hipFree(B_d);
    hipFree(C_d);
    return 0;

}

