#include <hip/hip_runtime.h>
#include <iostream>

__global__ void copyDataNonCoalesced(float *in, float *out, int n) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < n) {
        out[index] = in[(index * 2) % n];
    }
}

__global__ void copyDataCoalesced(float *in, float *out, int n) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < n) {
        out[index] = in[index];
    }
}

void initializeArray(float *arr, int n) {
    for(int i = 0; i < n; ++i) {
        arr[i] = static_cast<float>(i);
    }
}

int main() {
    const int n = 1 << 24; // Increase n to have a larger workload
    float *in, *out;
    hipEvent_t start, stop;
    float milliseconds = 0;
    
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipMallocManaged(&in, n * sizeof(float));
    hipMallocManaged(&out, n * sizeof(float));

    initializeArray(in, n);
    
    // Ensure data is on device before timing
    hipDeviceSynchronize();

    // int blockSize = 128; // Define block size
    int blockSize = 1024;
    int numBlocks = (n + blockSize - 1) / blockSize;

    // Time non-coalesced kernel
    hipEventRecord(start);
    copyDataNonCoalesced<<<numBlocks, blockSize>>>(in, out, n);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    std::cout << "Non-coalesced kernel time: " << milliseconds << " ms" << std::endl;

    // Reset output array
    initializeArray(out, n);
    hipDeviceSynchronize();

    // Time coalesced kernel
    hipEventRecord(start);
    copyDataCoalesced<<<numBlocks, blockSize>>>(in, out, n);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    std::cout << "Coalesced kernel time: " << milliseconds << " ms" << std::endl;

    // Cleanup
    hipEventDestroy(start);
    hipEventDestroy(stop);
    hipFree(in);
    hipFree(out);

    return 0;
}