/******************************************************************************
 *cr
 *cr            (C) Copyright 2010 The Board of Trustees of the
 *cr                        University of Illinois
 *cr                         All Rights Reserved
 *cr
 ******************************************************************************/


#include <hip/hip_runtime.h>
#include <cinttypes>
#include <stdio.h>

__global__ void mysgemm(int m, int n, int k, const float *A, const float *B, float* C) {

    /********************************************************************
     *
     * Compute C = A x B
     *   where A is a (m x k) matrix
     *   where B is a (k x n) matrix
     *   where C is a (m x n) matrix
     *
     ********************************************************************/

    // INSERT KERNEL CODE HERE
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < m && col < n) {
        float sum = 0;
        for (int i = 0; i < k; i++) {
            sum += A[row * k + i] * B[i * n + col];
        }
        C[row * n + col] = sum;
    }
}

__global__ void mysgemm_v2(int m, int n, int k, const float *A, const float *B, float* C) {

    /********************************************************************
     *
     * Compute C = A x B
     *   where A is a (m x k) matrix
     *   where B is a (k x n) matrix
     *   where C is a (m x n) matrix
     *
     ********************************************************************/

    // INSERT KERNEL CODE HERE
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    if (row  < m && col < n) {
        for (int i = 0; i < n; i++) {
            float sum = 0;
            for (int j = 0; j < k; j++) {
                sum += A[row * k + j] * B[j * n + i];
            }
            C[row * n + i] = sum;
        }
    }

}


void basicSgemm(char transa, char transb, int m, int n, int k, float alpha, const float *A, int lda, const float *B, int ldb, float beta, float *C, int ldc)
{
    if ((transa != 'N') && (transa != 'n')) {
	printf("unsupported value of 'transa'\n");
    	return;
    }

    if ((transb != 'N') && (transb != 'n')) {
	printf("unsupported value of 'transb'\n");
	return;
    }

    if ((alpha - 1.0f > 1e-10) || (alpha - 1.0f < -1e-10)) {
	printf("unsupported value of alpha\n");
	return;
    }

    if ((beta - 0.0f > 1e-10) || (beta - 0.0f < -1e-10)) {
	printf("unsupported value of beta\n");
	return;
    }

    // Initialize thread block and kernel grid dimensions ---------------------

    // const unsigned int BLOCK_SIZE = 16; // Use 16x16 thread blocks
    const unsigned int BLOCK_SIZE = 32; // Use 1x32 thread blocks
    //INSERT CODE HERE
    dim3 dim_grid, dim_block;
    dim_block = dim3(1, BLOCK_SIZE, 1);
    dim_grid = dim3(n, (m + BLOCK_SIZE - 1) / BLOCK_SIZE, 1);

    // Invoke CUDA kernel -----------------------------------------------------

    //INSERT CODE HERE
    mysgemm<<<dim_grid, dim_block>>>(m, n, k, A, B, C);

}


